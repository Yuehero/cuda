#include "hip/hip_runtime.h"
#pragma once
#include"cuda_helper.cuh"


#define MAX_NUM_DEVICES (1)
#define NUM_ELEM (1024*1024)
#define FULL_DATA_SIZE (NUM_ELEM*12)

// define one stream per GPU
__host__ void fillHostArray(u32* data, const u32 num_elements){
	for (u32 i = 0; i < num_elements; i++)
		data[i] = rand() % (num_elements + 1);
}
__host__ void checkArray(char *device_perfix, u32*data, const u32 num_elements){
	bool error = false;
	for (u32 i = 0; i < num_elements; i++)
		if (data[i] != i * 2){
			printf("%s error: %u %u", device_perfix, i, data[i]);
			error = true;
		}
	if (error == false){
		printf("%s array check passed.", device_perfix);
	}
}
__global__ void gpuKernel(u32 *data){
	int tid = threadIdx.x + blockDim.x*blockIdx.x;
	data[tid] *= 2;
}
void _streamTest_0(){

	hipStream_t stream[MAX_NUM_DEVICES];
	char device_perfix[MAX_NUM_DEVICES][300];
	u32*gpu_data[MAX_NUM_DEVICES];
	u32*cpu_src_data[MAX_NUM_DEVICES];
	u32*cpu_dest_data[MAX_NUM_DEVICES];

	//hipEvent_t kernel_start_event[MAX_NUM_DEVICES];
	//hipEvent_t mempcy_to_start_event[MAX_NUM_DEVICES];
	//hipEvent_t mempcy_from_start_event[MAX_NUM_DEVICES];
	//hipEvent_t mempcy_from_stop_event[MAX_NUM_DEVICES];

	//float time_kernel_ms[MAX_NUM_DEVICES];
	//float time_copy_to_ms[MAX_NUM_DEVICES];
	//float time_copy_from_ms[MAX_NUM_DEVICES];
	//float time_exec_ms[MAX_NUM_DEVICES];


	const int shared_memory_usage = 0;
	const size_t single_gpu_chunk_size = sizeof(u32)*NUM_ELEM;

	const u32 threadsPerBlock = 256;
	const u32 blocksPerGrid = (NUM_ELEM + threadsPerBlock - 1) / threadsPerBlock;

	int num_devices;
	checkCudaErrors(hipGetDeviceCount(&num_devices));
	if (num_devices > MAX_NUM_DEVICES)
		num_devices = MAX_NUM_DEVICES;

	for (int cur_device = 0; cur_device < num_devices; cur_device++){
		//select correct devices
		checkCudaErrors(hipSetDevice(cur_device));
		struct hipDeviceProp_t device_prop;
		checkCudaErrors(hipGetDeviceProperties(&device_prop, cur_device));
		printf(&device_perfix[cur_device][0], "\nID:%d %s", cur_device, device_prop.name);

		//create a stream
		checkCudaErrors(hipStreamCreate(&stream[cur_device]));
		//allocate device memory
		checkCudaErrors(hipMalloc(&gpu_data[cur_device], single_gpu_chunk_size));
		//allocate host memory
		checkCudaErrors(hipHostAlloc(&cpu_src_data[cur_device], single_gpu_chunk_size, hipHostMallocDefault));
		checkCudaErrors(hipHostAlloc(&cpu_dest_data[cur_device], single_gpu_chunk_size, hipHostMallocDefault));

		fillHostArray(cpu_src_data[cur_device], NUM_ELEM);

		//asynchronous
		//checkCudaErrors(hipEventRecord(mempcy_to_start_event[cur_device], 0));
		checkCudaErrors(hipMemcpyAsync(gpu_data[cur_device], cpu_src_data[cur_device], single_gpu_chunk_size, hipMemcpyHostToDevice, stream[cur_device]));

		//checkCudaErrors(hipEventRecord(kernel_start_event[cur_device], stream[cur_device]));
		gpuKernel << <blocksPerGrid, threadsPerBlock, shared_memory_usage, stream[cur_device] >> >(gpu_data[cur_device]);

		//checkCudaErrors(hipEventRecord(mempcy_from_start_event[cur_device], stream[cur_device]));
		checkCudaErrors(hipMemcpyAsync(cpu_dest_data[cur_device], gpu_data[cur_device], single_gpu_chunk_size, hipMemcpyDeviceToHost, stream[cur_device]));

		//checkCudaErrors(hipEventRecord(mempcy_from_stop_event[cur_device], 0));
	}
	//free
	for (int cur_device = 0; cur_device < num_devices; cur_device++){
		//select the correct device
		checkCudaErrors(hipSetDevice(cur_device));
		//wait for all commands in stream is complete
		checkCudaErrors(hipStreamSynchronize(stream[cur_device]));

		//checkCudaErrors(hipEventElapsedTime(&time_copy_to_ms[cur_device], mempcy_to_start_event[cur_device], kernel_start_event[cur_device]));

		//checkCudaErrors(hipEventElapsedTime(&time_kernel_ms[cur_device], kernel_start_event[cur_device], mempcy_from_start_event[cur_device]));

		//checkCudaErrors(hipEventElapsedTime(&time_copy_from_ms[cur_device], mempcy_from_start_event[cur_device], mempcy_from_stop_event[cur_device]));

		//checkCudaErrors(hipEventElapsedTime(&time_exec_ms[cur_device], mempcy_to_start_event[cur_device], mempcy_from_stop_event[cur_device]));

		//printf("> %s Copy to  \t:%.2f ms", device_perfix[cur_device], time_copy_to_ms[cur_device]);

		//printf("> %s Kernel   \t:%.2f ms", device_perfix[cur_device], time_kernel_ms[cur_device]);

		//printf("> %s Copy Back\t:%.2f ms", device_perfix[cur_device], time_copy_from_ms[cur_device]);

		//printf("> %s Execution \t:%.2f ms", device_perfix[cur_device], time_exec_ms[cur_device]);

		//checkArray(device_perfix[cur_device],cpu_dest_data[cur_device],NUM_ELEM);

		checkCudaErrors(hipStreamDestroy(stream[cur_device]));
		checkCudaErrors(hipFree(gpu_data[cur_device]));
		checkCudaErrors(hipHostFree(cpu_src_data[cur_device]));
		checkCudaErrors(hipHostFree(cpu_dest_data[cur_device]));


		//checkCudaErrors(hipEventDestroy(kernel_start_event[cur_device]));
		//checkCudaErrors(hipEventDestroy(mempcy_to_start_event[cur_device]));
		//checkCudaErrors(hipEventDestroy(mempcy_from_start_event[cur_device]));
		//checkCudaErrors(hipEventDestroy(mempcy_from_stop_event[cur_device]));
		hipDeviceReset();
	}
}

__global__ void gpuKernel_1(const u32 *src_arr0, const u32* src_arr1, u32 *const dst_arr){
	int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if (tid < NUM_ELEM){
		register u32 idx0 = (tid + 1) % 256;
		register u32 idx1 = (tid + 2) % 256;
		register float as = (src_arr0[tid] + src_arr0[idx0] + src_arr0[idx1]) / 3.0f;
		register float bs = (src_arr1[tid] + src_arr1[idx0] + src_arr1[idx1]) / 3.0f;
		dst_arr[tid] = (as + bs) / 2;
	}

}
void _multipleStreamTest(){

	int num_devices;
	checkCudaErrors(hipGetDeviceCount(&num_devices));

	printf("> GPU num: %d.\n", num_devices);

	int cur_device = 0;
	//select correct devices
	checkCudaErrors(hipSetDevice(cur_device));
	struct hipDeviceProp_t device_prop;
	checkCudaErrors(hipGetDeviceProperties(&device_prop, cur_device));
	printf("> ID:%d %s.\n", cur_device, device_prop.name);

	if (!device_prop.deviceOverlap){
		printf("> Device will not handle overlaps, so no speed up from stream\n");
	}
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	u32 const nstream = 2;
	hipStream_t stream[nstream];
	for (u32 str = 0; str < nstream; str++)
		checkCudaErrors(hipStreamCreate(&stream[str]));

	u32 *h_src_arr0, *h_src_arr1, *h_dst_arr;
	u32 *d_src_arr0[nstream], *d_src_arr1[nstream], *d_dst_arr[nstream];

	const u32 threadsPerBlock = 512;
	const u32 blocksPerGrid = (NUM_ELEM + threadsPerBlock - 1) / threadsPerBlock;

	const int shared_memory_usage = 0;
	const size_t Hsize = sizeof(u32)*FULL_DATA_SIZE;
	const size_t Dsize = sizeof(u32)*NUM_ELEM;

	checkCudaErrors(hipHostAlloc(&h_src_arr0, Hsize, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc(&h_src_arr1, Hsize, hipHostMallocDefault));
	checkCudaErrors(hipHostAlloc(&h_dst_arr, Hsize, hipHostMallocDefault));

	for (int str = 0; str < nstream; str++){
		checkCudaErrors(hipMalloc(&d_src_arr0[str], Dsize));
		checkCudaErrors(hipMalloc(&d_src_arr1[str], Dsize));
		checkCudaErrors(hipMalloc(&d_dst_arr[str], Dsize));
	}

	fillHostArray(h_src_arr0, FULL_DATA_SIZE);
	fillHostArray(h_src_arr1, FULL_DATA_SIZE);

	checkCudaErrors(hipEventRecord(start, 0));

	for (int i = 0; i < FULL_DATA_SIZE; i += NUM_ELEM * nstream){

		for (int str = 0; str < nstream; str++){
			checkCudaErrors(hipMemcpyAsync(d_src_arr0[str], h_src_arr0 + i + NUM_ELEM*str, Dsize, hipMemcpyHostToDevice, stream[str]));
			checkCudaErrors(hipMemcpyAsync(d_src_arr1[str], h_src_arr1 + i + NUM_ELEM*str, Dsize, hipMemcpyHostToDevice, stream[str]));

			gpuKernel_1 << <blocksPerGrid, threadsPerBlock, shared_memory_usage, stream[str] >> >
				(d_src_arr0[str], d_src_arr1[str], d_dst_arr[str]);

			checkCudaErrors(hipMemcpyAsync(h_dst_arr + i + NUM_ELEM*str, d_dst_arr[str], Dsize, hipMemcpyDeviceToHost, stream[str]));
		}
	}

	//for (int i = 0; i < FULL_DATA_SIZE; i += NUM_ELEM * nstream){

	//	checkCudaErrors(hipMemcpyAsync(d_src_arr0[0], h_src_arr0 + i, Dsize, hipMemcpyHostToDevice, stream[0]));
	//	checkCudaErrors(hipMemcpyAsync(d_src_arr1[0], h_src_arr1 + i, Dsize, hipMemcpyHostToDevice, stream[0]));

	//	checkCudaErrors(hipMemcpyAsync(d_src_arr0[1], h_src_arr0 + i + NUM_ELEM, Dsize, hipMemcpyHostToDevice, stream[1]));
	//	checkCudaErrors(hipMemcpyAsync(d_src_arr1[1], h_src_arr1 + i + NUM_ELEM, Dsize, hipMemcpyHostToDevice, stream[1]));

	//	gpuKernel_1 << <blocksPerGrid, threadsPerBlock, shared_memory_usage, stream[0] >> >
	//		(d_src_arr0[0], d_src_arr1[0], d_dst_arr[0]);

	//	checkCudaErrors(hipMemcpyAsync(h_dst_arr + i, d_dst_arr[0], Dsize, hipMemcpyDeviceToHost, stream[0]));

	//	gpuKernel_1 << <blocksPerGrid, threadsPerBlock, shared_memory_usage, stream[1] >> >
	//		(d_src_arr0[1], d_src_arr1[1], d_dest_arr[1]);

	//	checkCudaErrors(hipMemcpyAsync(h_dst_arr + i + NUM_ELEM, d_dst_arr[1], Dsize, hipMemcpyDeviceToHost, stream[1]));

	//}
	for (int str = 0; str < nstream; str++)
		checkCudaErrors(hipStreamSynchronize(stream[str]));
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));

	float elapsedTime = 0.f;
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("> Time taken: %3.1f ms\n", elapsedTime);

	checkCudaErrors(hipHostFree(h_src_arr0));
	checkCudaErrors(hipHostFree(h_src_arr1));
	checkCudaErrors(hipHostFree(h_dst_arr));
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	for (int str = 0; str < nstream; str++){
		checkCudaErrors(hipStreamDestroy(stream[str]));
		checkCudaErrors(hipFree(d_src_arr0[str]));
		checkCudaErrors(hipFree(d_src_arr1[str]));
		checkCudaErrors(hipFree(d_dst_arr[str]));
	}

}


//int main(int argc, char **argv)
//{
//	printf("> %s Starting...\n\n", argv[0]);
//	srand((unsigned)time(NULL));
//
//	_multipleStreamTest();
//
//	exit(EXIT_SUCCESS);
//}